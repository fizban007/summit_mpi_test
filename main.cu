#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "mpi.h"

__global__ void set_mem(double* buffer, int size, double value) {
  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < size;
       n += gridDim.x * blockDim.x) {
    buffer[n] = value;
  }
}

int
main(int argc, char* argv[]) {
  MPI_Init(NULL, NULL);
  
  int rank, size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);

  std::cout << "rank is: " << rank << ", total size is: " << size << std::endl;

  int n_devices;
  hipGetDeviceCount(&n_devices);
  if (n_devices <= 0) {
    std::cerr << "No usable Cuda device found!!" << std::endl;
    exit(1);
  } else {
    std::cout << "Found " << n_devices << " Cuda devices!" << std::endl;
  }
  // set device number
  int dev_id = rank % n_devices;
  std::cout << "Rank " << rank << " is on device #" << dev_id << std::endl;
  hipSetDevice(dev_id);

  // Prepare the buffer for communication
  double* dev_send_buffer;
  double* dev_recv_buffer;
  
  int N = 1000000;
  hipMalloc(&dev_send_buffer, N * sizeof(double));
  hipMalloc(&dev_recv_buffer, N * sizeof(double));

  set_mem<<<512, 512>>>(dev_send_buffer, N, (double)rank);
  hipDeviceSynchronize();

  // Send the buffer to the next rank
  int dst = (rank + 1) % size;
  int src = (rank - 1 + size) % size;
  std::cout << "Sending to rank " << dst << ", receiving from rank " << src << std::endl;

  for (int n = 0; n < 100; n++) {
    size_t free_mem, total_mem;
    MPI_Status status;
    MPI_Sendrecv(dev_send_buffer, N, MPI_DOUBLE, dst, 0,
                dev_recv_buffer, N, MPI_DOUBLE, src, 0,
                MPI_COMM_WORLD, &status);
    hipMemGetInfo( &free_mem, &total_mem );
    std::cout << "GPU memory: free=" << free_mem/1.0e9 << "GiB, total=" << total_mem/1.0e9 << "GiB" << std::endl;
  }

  std::vector<double> host_recv_buffer(N);
  hipMemcpy(host_recv_buffer.data(), dev_recv_buffer, N * sizeof(double), hipMemcpyDeviceToHost);
  std::cout << "value is " << host_recv_buffer[N - 1] << " on rank " << rank << std::endl;
  
  hipFree(dev_send_buffer);
  hipFree(dev_recv_buffer);

  MPI_Finalize();
  
  return 0;
}
